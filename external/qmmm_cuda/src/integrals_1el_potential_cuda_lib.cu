#include "hip/hip_runtime.h"
/* The GPU Optimized QMMM Library, version 0.1, a Portable GPU library for evaluation of Coulomb interaction
 * integrals between electrons in QM region of the system and point charges in MM
 * region of the system. This library is designed to accelerate hybrid QM/MM computations
 * in quantum chemistry programs. It can be integrated into any quantum chemistry program,
 * which supports QM/MM computations, via standard C interface.
 *
 * Copyright (C) 2013 Mikael Engbom
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 * For further information about The GPU Optimized Library, see http://www.scalalife.eu/content/portable-gpu-libary-qm-mm-calculations
 */

/* The GPU optimized QMMM library, version 0.1, is based on the Ergo 3.1 source code */

/* Ergo, version 3.1, a program for linear scaling electronic structure
 * calculations.
 * Copyright (C) 2011 Elias Rudberg, Emanuel H. Rubensson, and Pawel Salek.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 * Primary academic reference:
 * Kohn−Sham Density Functional Theory Electronic Structure Calculations
 * with Linearly Scaling Computational Time and Memory Usage,
 * Elias Rudberg, Emanuel H. Rubensson, and Pawel Salek,
 * J. Chem. Theory Comput. 7, 340 (2011),
 * <http://dx.doi.org/10.1021/ct100611z>
 *
 * For further information about Ergo, see <http://www.ergoscf.org>.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <limits>
#include <cmath>
#include <vector>
#include <iomanip>
#include <omp.h>

#include "qmmm_utility.h"
#include "integrals_1el_potential_cuda_lib.h"

#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif

using namespace std;

struct __align__(16) cuda_atom_upper {
	qmmm_real charge;
	qmmm_real coords_x;
};

struct __align__(16) cuda_atom_lower {
	qmmm_real coords_y;
	qmmm_real coords_z;
};

struct cuda_atom {
	cuda_atom_upper upper;
	cuda_atom_lower lower;
};

struct cuda_boys_func_interval_struct {
	qmmm_real midx;
	double2 a[QMMM_BOYS_TAB_DEGREE / 2];
};

__constant__ int d_number_of_atoms;
__constant__ cuda_atom* d_cuda_atoms;
__constant__ cuda_boys_func_interval_struct* d_cuda_boys_list;

static bool isPow2(unsigned int x) {
	return ((x&(x-1))==0);
}

template <unsigned int blockSize, bool nIsPow2>
__global__ void reduce_kernel(double *d_in, double *d_out, unsigned int n) {
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
	unsigned int gridSize = blockSize*2*gridDim.x;

	extern __shared__ double smem[];

	double sum = 0.0f;

	while (i < n) {
		sum += d_in[i];
		if (nIsPow2 || i + blockSize < n) {
			sum += d_in[i+blockSize];
		}
		i += gridSize;
	}

	smem[tid] = sum;
	__syncthreads();


	if (blockSize >= 512) { if (tid < 256) { smem[tid] = sum = sum + smem[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { smem[tid] = sum = sum + smem[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid <  64) { smem[tid] = sum = sum + smem[tid +  64]; } __syncthreads(); }

	if (tid < 32) {
		volatile double* smemv = smem;
		if (blockSize >=  64) { smemv[tid] = sum = sum + smemv[tid + 32]; __syncthreads(); }
		if (blockSize >=  32) { smemv[tid] = sum = sum + smemv[tid + 16]; __syncthreads(); }
		if (blockSize >=  16) { smemv[tid] = sum = sum + smemv[tid +  8]; __syncthreads(); }
		if (blockSize >=   8) { smemv[tid] = sum = sum + smemv[tid +  4]; __syncthreads(); }
		if (blockSize >=   4) { smemv[tid] = sum = sum + smemv[tid +  2]; __syncthreads(); }
		if (blockSize >=   2) { smemv[tid] = sum = sum + smemv[tid +  1]; __syncthreads(); }
	}

	if (tid == 0) {
		d_out[blockIdx.x] = smem[0];
	}
}

// Wrapper function for kernel launch
void reduce(int size, int threads, int blocks, double *d_in, double *d_out, hipStream_t stream) {

	int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

	if (isPow2(size)) {
		switch (threads) {
		case 512:
			reduce_kernel<512, true><<< blocks, threads, smemSize, stream >>>(d_in, d_out, size); break;
		case 256:
			reduce_kernel<256, true><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case 128:
			reduce_kernel<128, true><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case 64:
			reduce_kernel<64, true><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case 32:
			reduce_kernel<32, true><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case 16:
			reduce_kernel<16, true><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case  8:
			reduce_kernel<8, true><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case  4:
			reduce_kernel<4, true><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case  2:
			reduce_kernel<2, true><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case  1:
			reduce_kernel<1, true><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		}
	}
	else {
		switch (threads) {
		case 512:
			reduce_kernel<512, false><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case 256:
			reduce_kernel<256, false><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case 128:
			reduce_kernel<128, false><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case 64:
			reduce_kernel<64, false><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case 32:
			reduce_kernel<32, false><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case 16:
			reduce_kernel<16, false><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case  8:
			reduce_kernel<8, false><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case  4:
			reduce_kernel<4, false><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case  2:
			reduce_kernel<2, false><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		case  1:
			reduce_kernel<1, false><<< blocks, threads, smemSize,stream >>>(d_in, d_out, size); break;
		}
	}
}


/* To reduce number of register, center_arguments are passed into variables dx, dy, dz */
__global__ void compute_V_matrix_full_gpu_kernel_0(qmmm_real dx,
		qmmm_real dy,
		qmmm_real dz,
		const qmmm_real alpha,
		const qmmm_real boys_pre_factor,
		const qmmm_real factor,
		qmmm_real* d_result) {

	/* loop over atoms */
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	qmmm_real r_start;

	if(i < d_number_of_atoms) {
		cuda_atom atom = d_cuda_atoms[i];

		dx = atom.upper.coords_x - dx;
		dy = atom.lower.coords_y - dy;
		dz = atom.lower.coords_z - dz;

		const qmmm_real dxyz_squared = dx*dx + dy*dy + dz*dz;
		const qmmm_real arg = alpha * dxyz_squared;

		if(arg  >= QMMM_BOYS_X_MAX) {
			r_start = boys_pre_factor * rsqrt(arg);
		} else {
			/* choose which interval to use */
			const int intervalIndex = (int) arg * (qmmm_real) (QMMM_BOYS_NO_OF_INTERVALS / QMMM_BOYS_X_MAX);

			const cuda_boys_func_interval_struct* interval = &d_cuda_boys_list[intervalIndex];
			const qmmm_real deltax = arg - (qmmm_real)QMMM_BOYS_TEMP * (intervalIndex + 0.5);// arg - interval->midx;

			qmmm_real deltaxtopowk = 1;
			r_start = 0;

			for(int k = 0; k < QMMM_BOYS_TAB_DEGREE / 2; k++) {
				r_start += interval->a[k].x * deltaxtopowk;
				deltaxtopowk *= deltax;

				r_start += interval->a[k].y * deltaxtopowk;
				deltaxtopowk *= deltax;
			}
		}

		d_result[i] =  atom.upper.charge * factor * r_start;
	} // END IF ATOMS < NUMBER OF ATOMS
}

__global__ void compute_V_matrix_full_gpu_kernel(qmmm_real* d_result,
		const qmmm_real* d_work_list,
		int size) {

	/*  loop over atoms */
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	qmmm_real dx, dy, dz, dxyz_squared, alpha;

	qmmm_real arg, expMinusArg, r_start;

	int n_max;
	qmmm_real res = 0;
	qmmm_real tmp_xyz = 1;

	if(i < d_number_of_atoms) {
		cuda_atom atom = d_cuda_atoms[i];
		int j = 0;
		while (j < size) {
			switch ((int)d_work_list[j++]) {
			case NEW_CENTER :
				dx = atom.upper.coords_x - d_work_list[j++];
				dy = atom.lower.coords_y - d_work_list[j++];
				dz = atom.lower.coords_z - d_work_list[j++];
				dxyz_squared = dx*dx + dy*dy + dz*dz;
				break;
			case NEW_ALPHA :
				alpha = d_work_list[j++];
				arg = alpha * dxyz_squared;
				expMinusArg = std::exp(-arg);
				break;
			case  NEW_NMAX :
			{
				n_max = (int)d_work_list[j++];

				if(arg  >= QMMM_BOYS_X_MAX) {
					const qmmm_real boys_pre_factor = d_work_list[j];
					j++;
					qmmm_real arg_exp = 1;
					for(int k = 0; k < 2*n_max + 1; k++) {
						arg_exp *= arg;
					}
					r_start = boys_pre_factor * rsqrt(arg_exp);
				} else {
					j++; //skip boys_pre_factor

					/* choose which interval to use */
					const int intervalIndex = arg * (qmmm_real) (QMMM_BOYS_NO_OF_INTERVALS / QMMM_BOYS_X_MAX);

					const cuda_boys_func_interval_struct* interval = &d_cuda_boys_list[n_max*QMMM_BOYS_NO_OF_INTERVALS + intervalIndex];
					const qmmm_real deltax = arg - (qmmm_real)QMMM_BOYS_TEMP * (intervalIndex + 0.5);// arg - interval->midx;

					qmmm_real deltaxtopowk = 1;
					r_start = 0;

					for(int k = 0; k < QMMM_BOYS_TAB_DEGREE / 2; k++) {
						r_start += interval->a[k].x * deltaxtopowk;
						deltaxtopowk *= deltax;

						r_start += interval->a[k].y * deltaxtopowk;
						deltaxtopowk *= deltax;
					}
				}
				break;
			}
			case ADD :
			{
				const qmmm_real factor = d_work_list[j++];
				const int n = (int)d_work_list[j++];

				qmmm_real r = r_start;

				// get boys factor
				if(n_max > 0) {
					for(int k = n_max-1; k >= n; k--)
						r = (2*arg*r + expMinusArg) / (2*k+1);
				}

				res += tmp_xyz * factor * r;
				tmp_xyz = 1;
				break;
			}
			case DX :
				tmp_xyz *= dx;
				break;
			case DY :
				tmp_xyz *= dy;
				break;
			case DZ :
				tmp_xyz *= dz;
				break;
			}
		}
		d_result[i] =  atom.upper.charge * res;
	} // END IF ATOMS < NUMBER OF ATOMS
}

class monomial_help_class {
public:
	qmmm_real factor;
	int length;
	int x;
	int y;
	int z;

	monomial_help_class(qmmm_real factor, int length, int x, int y, int z)
	:factor(factor), length(length), x(x), y(y), z(z)
	{};

	std::vector<monomial_help_class> operator*(std::vector<monomial_help_class> rhs);
};

std::vector<monomial_help_class> monomial_help_class::operator*(std::vector<monomial_help_class>  rhs) {
	std::vector<monomial_help_class> m;

	for(unsigned int i = 0; i < rhs.size(); i++) {
		m.push_back(monomial_help_class(factor * rhs[i].factor, length + rhs[i].length, x + rhs[i].x, y + rhs[i].y, z + rhs[i].z));
	}
	return m;
}


static unsigned int nextPow2( unsigned int x ) {
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}

static vector<int> get_compatible_devices() {

	unsigned int cuda_success = 0;

	int number_of_GPUs;
	cuda_success |= (unsigned int)(hipGetDeviceCount(&number_of_GPUs));
	cout << "Found " << number_of_GPUs << " GPUs (cuda_success = " << cuda_success << ")" << endl;
	vector<int> devices;
	for(int i = 0; i < number_of_GPUs; i++) {
		int dev_ID;
		hipDeviceProp_t props;
		cuda_success |= (unsigned int) hipGetDevice(&dev_ID);
		cuda_success |= (unsigned int) hipGetDeviceProperties(&props, i);
		printf("Device %d: \"%s\" with Compute %d.%d capability found\n", dev_ID, props.name, props.major, props.minor);
		//if(props.major == 2 && props.minor == 0) {
		if(props.major >= 2) {
			devices.push_back(i);
		}
	}

	if(cuda_success != 0) {
		cout << "[Function get_compatible_devices] CUDA error = " << cuda_success << endl;
	}
	return devices;
}


static void get_num_blocks_and_threads(int n, int maxBlocks, int maxThreads, int &blocks, int &threads) {
	threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
	blocks = (n + (threads * 2 - 1)) / (threads * 2);
	blocks = MIN(maxBlocks, blocks);
}

static vector<monomial_help_class>  genereate_polynom(int n, const int x, const int y, const int z) {

	vector<monomial_help_class> monomial_vector;
	if(x == 0 && y == 0 && z == 0) {
		monomial_help_class m(1, n, 0, 0, 0);
		vector<monomial_help_class> m_v;
		m_v.push_back(m);
		return m_v;
	}

	if(x > 0) {
		monomial_vector = monomial_help_class(1, 0, 1, 0, 0) * genereate_polynom(n +1, x-1 ,y, z);
		if(x > 1) {
			vector<monomial_help_class> vec_tmp = monomial_help_class(x-1, 0, 0, 0, 0) * genereate_polynom(n +1, x-2, y, z);
			monomial_vector.insert(monomial_vector.end(), vec_tmp.begin(), vec_tmp.end());
		}
	} else if(y > 0) {
		monomial_vector = monomial_help_class(1, 0, 0, 1, 0) * genereate_polynom(n +1, x, y-1, z);
		if(y > 1) {
			vector<monomial_help_class> vec_tmp = monomial_help_class(y-1, 0, 0, 0, 0) * genereate_polynom(n +1, x, y-2, z);
			monomial_vector.insert(monomial_vector.end(), vec_tmp.begin(), vec_tmp.end());
		}
	} else if(z > 0) {
		monomial_vector = monomial_help_class(1, 0, 0, 0, 1) * genereate_polynom(n +1, x, y, z-1);
		if(z > 1) {
			vector<monomial_help_class> vec_tmp = monomial_help_class(z-1, 0, 0, 0, 0) * genereate_polynom(n +1, x, y, z-2);
			monomial_vector.insert(monomial_vector.end(), vec_tmp.begin(), vec_tmp.end());
		}
	}
	return monomial_vector;
}

static int clean_work_list(vector <double> &list) {
	/* Remove ADD with factor == 0 (will result in multiply with 0 => result = 0 (this is done in the collection code)
	 * Remove NEW_NMAX if new NEW_MAX is found without an ADD between
	 * Remove NEW ALPHA if new NEW_ALPHA is found without an NEW_NMAX is found between
	 * Remove NEW_CENTERCOORDS if NEW_CENTERCOORDS is found without an NEW_NMAX is found between
	 * */
	int old_alpha_index = -1;
	int old_n_max_index = -1;
	bool n_max_found = false;
	bool add_found = true;
	bool center_found = true;

	int i = 0;
	int size = list.size();

	//Clean up N_MAX
	while (i < size) {
		switch ((int)list[i]) {
		case NEW_CENTER :
			i +=  4;
			break;
		case NEW_ALPHA :
			i += 2;
			break;
		case  NEW_NMAX :
			if(!add_found) {
				list.erase(list.begin() + old_n_max_index, list.begin() + old_n_max_index + 3);
				old_n_max_index = i-3;
				size = list.size();
			} else {
				old_n_max_index = i;
				i += 3;
			}
			add_found = false;
			break;
		case ADD :
			add_found = true;
			i += 3;
			break;
		default :
			i++;
			break;
		}
	}

	i = 0;
	size = list.size();
	n_max_found = true;
	//Clean up ALPHA
	while (i < size) {
		switch ((int)list[i]) {
		case NEW_CENTER :
			center_found = true;
			i += 4;
			break;
		case NEW_ALPHA :
			if(!n_max_found && !center_found) {
				cout << "Erasing an alpha" << endl;
				list.erase(list.begin() + old_alpha_index, list.begin() + old_alpha_index + 2);
				old_alpha_index = i-2;
				size = list.size();
			} else {
				old_alpha_index = i;
				i += 2;
			}
			center_found = false;
			n_max_found = false;
			break;
		case  NEW_NMAX :
			n_max_found = true;
			i += 3;
			break;
		case ADD :
			i += 3;
			break;
		default :
			i++;
			break;
		}
	}
	return 0;
}

static bool is_case_0(vector <double> &list) {
	bool case_0 = false;
	int i = 0;
	int size = list.size();
	int number_of_new_center = 0;
	int number_of_new_alpha = 0;
	int number_of_new_n_max = 0;
	int number_of_new_add = 0;
	int number_of_dx = 0;
	int sum_n_max = 0;
	int sum_length = 0;

	//Clean up N_MAX
	while (i < size) {
		switch ((int)list[i]) {
		case NEW_CENTER :
			number_of_new_center++;
			i = i + 4;
			break;
		case NEW_ALPHA :
			number_of_new_alpha++;
			i = i +2;
			break;
		case  NEW_NMAX :
			number_of_new_n_max++;
			sum_n_max += (int)list[i+1];
			i += 3;
			break;
		case ADD :
			number_of_new_add++;
			sum_length += (int)list[i+2];
			i += 3;
			break;
		default :
			number_of_dx++;
			i++;
			break;
		}
	}
	if(number_of_new_center == 1 && number_of_new_alpha == 1 && number_of_new_n_max == 1 && number_of_new_add == 1 && number_of_dx == 0 && sum_n_max == 0 && sum_length == 0) {
		case_0 = true;
	}
	return case_0;
}

// Library version (library input, cleaned version)
int compute_V_matrix_full_gpu_lib(const qmmm_basis_info_struct& basis_info,
		const int num_qm_atoms,
		const qmmm_atom* qm_atom_list,
		const int num_mm_atoms,
		const qmmm_atom* mm_atom_list,
		const qmmm_real threshold,
		qmmm_real* result) {

	const int number_of_basis_functions = basis_info.no_of_basis_funcs;
	const int iterations = (number_of_basis_functions * (number_of_basis_functions + 1)) / 2;
	int cntr[20];

	for(int i = 0; i < 20; i++) {
		cntr[i] = 0;
	}

	std::cout << "Enter compute_V_matrix_full_gpu" << std::endl;
	std::cout << "Number of basis functions = " << number_of_basis_functions << ",  number of qm atoms = " << num_qm_atoms << ", number of mm atoms " <<  num_mm_atoms << std::endl;
//	std::cout << "Number of iterations = " << iterations << std::endl;

#ifdef _OPENMP
	double time_start, time_stop;
	time_start = omp_get_wtime();
#endif

	const qmmm_monomial_info_struct* monomial_info = new qmmm_monomial_info_struct();
	const qmmm_hermite_conversion_info_struct* hermite_conversion_info =  new qmmm_hermite_conversion_info_struct();

	boys_func_interval_struct boys_list[QMMM_BOYS_N_MAX][QMMM_BOYS_NO_OF_INTERVALS];
	qmmm_boysfunction_init(boys_list);

	// We use "cuda_atom" to be able to do 128bit copy from global memory to each thread on the GPU
	cuda_atom* cuda_atoms = (cuda_atom*) malloc(sizeof(cuda_atom) * num_mm_atoms);
	for(int i = 0; i < num_mm_atoms; i++) {
		cuda_atoms[i].upper.charge = mm_atom_list[i].charge;
		cuda_atoms[i].upper.coords_x = mm_atom_list[i].coords[0];
		cuda_atoms[i].lower.coords_y = mm_atom_list[i].coords[1];
		cuda_atoms[i].lower.coords_z = mm_atom_list[i].coords[2];
	}

	// We use "cuda_boys_func_interval_struct" to be able to do 128bit copy from global memory to each thread on the GPU
	cuda_boys_func_interval_struct * cuda_boys_list = (cuda_boys_func_interval_struct*) malloc(sizeof(cuda_boys_func_interval_struct) * QMMM_BOYS_N_MAX * QMMM_BOYS_NO_OF_INTERVALS);
	for(int i = 0; i < QMMM_BOYS_N_MAX; i ++) {
		for(int j = 0; j < QMMM_BOYS_NO_OF_INTERVALS; j++) {
			cuda_boys_list[i*QMMM_BOYS_NO_OF_INTERVALS + j].midx  = boys_list[i][j].midx;
			for(int k = 0; k < QMMM_BOYS_TAB_DEGREE / 2; k++) {
				cuda_boys_list[i*QMMM_BOYS_NO_OF_INTERVALS + j].a[k].x = boys_list[i][j].a[2*k];
				cuda_boys_list[i*QMMM_BOYS_NO_OF_INTERVALS + j].a[k].y = boys_list[i][j].a[2*k+1];
			}
		}
	}

	unsigned int cuda_success = 0;

	vector<int> devices = get_compatible_devices();

	if(devices.size() == 0) {
		cout << "Error! No devices  with compute capabilyt >= 2 found!\n" << endl;
		free(cuda_atoms);
		free(cuda_boys_list);
		return 0;
	}

	const int nr_streams = 4;

#ifdef _OPENMP

#pragma omp parallel reduction(+:cuda_success) num_threads(devices.size())
	{
		int thread_id = omp_get_thread_num();
		hipSetDevice(devices[thread_id]);
#endif

		int dev_ID;
		cuda_success |= (unsigned int) hipGetDevice(&dev_ID);

		hipSetDeviceFlags(hipDeviceScheduleSpin);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);

		std::vector<qmmm_distribution_spec_struct*> psi_list;

		qmmm_real* h_result;
		h_result = (qmmm_real*)malloc(sizeof(qmmm_real)*iterations);

		int estimated_max_size_work_list = 100;
		qmmm_real* d_work_list;
		qmmm_real* h_work_list;

		//		std::cout << "Allocating "<< sizeof(qmmm_real) * estimated_max_size_work_list*nr_streams / (1024*1024) <<" MB (d_work_list) on the GPU" << std::endl;
		cuda_success |= (unsigned int) hipMalloc(&d_work_list, sizeof(qmmm_real) * estimated_max_size_work_list*nr_streams);
		cuda_success |= (unsigned int) hipHostMalloc(&h_work_list,sizeof(qmmm_real)*estimated_max_size_work_list*nr_streams);

		//		std::cout << "Allocating "<< sizeof(cuda_boys_func_interval_struct) * BOYS_N_MAX * BOYS_NO_OF_INTERVALS / (1024*1024) <<" MB (d_cuda_boys_list) on the GPU" << std::endl;
		cuda_boys_func_interval_struct* d_cuda_boys_list;
		cuda_success |= (unsigned int) hipMalloc(&d_cuda_boys_list, sizeof(cuda_boys_func_interval_struct) * QMMM_BOYS_N_MAX * QMMM_BOYS_NO_OF_INTERVALS);
		cuda_success |= (unsigned int) hipMemcpy(d_cuda_boys_list, cuda_boys_list, sizeof(cuda_boys_func_interval_struct) * QMMM_BOYS_N_MAX * QMMM_BOYS_NO_OF_INTERVALS, hipMemcpyHostToDevice);

		//		std::cout << "Allocating "<< sizeof(cuda_atom) * number_of_atoms / (1024*1024) <<" MB (d_cuda_atoms) on the GPU" << std::endl;
		cuda_atom* d_cuda_atoms;
		cuda_success |= (unsigned int) hipMalloc(&d_cuda_atoms, sizeof(cuda_atom) * num_mm_atoms);
		cuda_success |= (unsigned int) hipMemcpy(d_cuda_atoms, cuda_atoms, sizeof(cuda_atom) * num_mm_atoms, hipMemcpyHostToDevice);

		qmmm_real* partial_result;
		partial_result = (qmmm_real*) malloc (sizeof(qmmm_real) * num_mm_atoms);

		//		std::cout << "Allocating "<< sizeof(qmmm_real) * number_of_atoms / (1024*1024) <<" MB (d_partial_result) on the GPU" << std::endl;
		qmmm_real* d_partial_result;
		cuda_success |= (unsigned int) hipMalloc(&d_partial_result, sizeof(qmmm_real) * num_mm_atoms);

		//		std::cout << "Allocating "<< sizeof(qmmm_real) * iterations / (1024*1024) << " MB (d_result) on the GPU" << std::endl;
		qmmm_real* d_result;
		cuda_success |= (unsigned int) hipMalloc(&d_result, sizeof(qmmm_real) * iterations);
		cuda_success |= hipMemset(d_result,0,iterations*sizeof(qmmm_real));

		//create event for sync.
		int current_stream = 0;
		hipEvent_t done[nr_streams];
		hipStream_t stream[nr_streams];
		for(int i=0; i < nr_streams; i++)
			cuda_success |= hipEventCreate(&done[i]);

		for (int i = 0; i < nr_streams; ++i)
			hipStreamCreate(&stream[i]);

		const int threadsPerBlock = 64;//256;
		const int blocksPerGrid = (num_mm_atoms + threadsPerBlock -1) / threadsPerBlock;

		int threadsInReduction, threadsInReduction2;
		int blocksInReduction, blocksInReduction2;
		const int maxThreads = 512;
		const int maxBlocks = 65535;

		get_num_blocks_and_threads(num_mm_atoms, maxBlocks, maxThreads, blocksInReduction, threadsInReduction);
		get_num_blocks_and_threads(blocksInReduction, maxBlocks, maxThreads, blocksInReduction2, threadsInReduction2);

		qmmm_real* partial_reduce_result = (qmmm_real*) malloc(sizeof(qmmm_real)*blocksInReduction);

		//		cout << "blocksInReduction = " << blocksInReduction << ", threadsInReduction = " << threadsInReduction << endl;
		//		cout << "blocksInReduction2 = " << blocksInReduction2 << ", threadsInReduction2 = " << threadsInReduction2 << endl;

		cuda_success |= hipMemcpyToSymbol(HIP_SYMBOL("d_number_of_atoms"), &num_mm_atoms, sizeof(int));
		cuda_success |= hipMemcpyToSymbol(HIP_SYMBOL("d_cuda_atoms"), &d_cuda_atoms, sizeof(cuda_atom*));
		cuda_success |= hipMemcpyToSymbol(HIP_SYMBOL("d_cuda_boys_list"), &d_cuda_boys_list, sizeof(cuda_boys_func_interval_struct*));

		//trigger event
		for(int i=0; i < nr_streams; i++) {
			hipEventRecord(done[i],stream[i]);
		}

		vector<int> iteration_indexes;
		/* This loop replaces mu and nu */
#ifdef _OPENMP
#pragma omp for schedule(dynamic, 800)
#endif
		for(int i = 0; i < iterations; i++) {

			const int mu = sqrt(2*i + 0.25) - 0.5;
			const int nu = i - (mu * (mu + 1)) / 2;
			const int result_index = mu*number_of_basis_functions + nu;

			iteration_indexes.push_back(i);

			//Get first contracted gaussian
			int start_prim_mu = basis_info.start_index[mu];
			int stop_prim_mu = basis_info.start_index[mu+1];

			//Get second contracted gaussian
			int start_prim_nu = basis_info.start_index[nu];
			int stop_prim_nu = basis_info.start_index[nu+1];

			/* compute matrix element [mu,nu] */
			qmmm_real old_alpha = -1;
			qmmm_real old_coord_x = -1;
			qmmm_real old_coord_y = -1;
			qmmm_real old_coord_z = -1;

			qmmm_real* center_coords;
			qmmm_real alpha;

			vector<qmmm_real> work_list;
			int old_n_max = -1;

			for(int j = start_prim_mu; j < stop_prim_mu; j++) {
				for(int k = start_prim_nu; k < stop_prim_nu; k++) {
					psi_list.clear();
					get_product_simple_prims(basis_info.simple_primitive_list[j], basis_info.simple_primitive_list[k], psi_list, threshold);

					if(psi_list.size() > 0) {
						// Alpha and centerCoords is same for all new prims
						center_coords = psi_list[0]->center_coords;
						alpha = psi_list[0]->exponent;
						const qmmm_real inv_alpha = 1 / alpha;
						const qmmm_real resultPreFactor = 2 * pi * inv_alpha;

						//Is center_corrds and alpha from last j, k same as this?
						if(center_coords[0] != old_coord_x || center_coords[1] != old_coord_y || center_coords[2] != old_coord_z) {
							work_list.push_back(NEW_CENTER);
							work_list.push_back(center_coords[0]);
							work_list.push_back(center_coords[1]);
							work_list.push_back(center_coords[2]);
							old_coord_x = center_coords[0];
							old_coord_y = center_coords[1];
							old_coord_z = center_coords[2];
							old_n_max = -1;
							old_alpha = -1; // need to add alpha after new center coords so that arg gets updated correctly
						}

						if(alpha != old_alpha) {
							work_list.push_back((NEW_ALPHA));
							work_list.push_back(alpha);
							old_alpha = alpha;
							old_n_max = -1;
						}

						for(int m = 0; m < psi_list.size(); m++) {
							const qmmm_distribution_spec_struct* psi = psi_list[m];

							const int n1x = psi->monomialInts[0];
							const int n1y = psi->monomialInts[1];
							const int n1z = psi->monomialInts[2];

							const int n1_max = n1x + n1y + n1z;
							const int n2_max = 0;
							const int n_max = n1_max + n2_max;

							if(n_max >= QMMM_BOYS_N_MAX) {
								cout << "Error, n_max >= BOYS_N_MAX!" << endl;
								exit(0);
							}

							if(old_n_max != n_max) {
								const qmmm_real boys_pre_factor_value = qmmm_boys_pre_factor(n_max);
								work_list.push_back(NEW_NMAX);
								work_list.push_back(n_max);
								work_list.push_back(boys_pre_factor_value);
								old_n_max = n_max;
							}

							const int monomialIndex = monomial_info->monomial_index_list[n1x][n1y][n1z];
							const int no_of_monomials_n1 = monomial_info->no_of_monomials_list[n1_max];
							const int no_of_monomials_n2 = monomial_info->no_of_monomials_list[n2_max];

							const int no_of_contribs = hermite_conversion_info->counters_right[n1_max][n2_max];
							const qmmm_hermite_conversion_contrib_struct* list = hermite_conversion_info->list_right[n1_max][n2_max];

							qmmm_real pre_factor;
							int x, y, z, sum_n, i1, i2;
							for(int n=0; n < no_of_contribs; n++) {
								if(list[n].destIndex == monomialIndex) {
									if(no_of_monomials_n1 > 0 && no_of_monomials_n2 > 0) {

										i1 = list[n].sourceIndex / no_of_monomials_n2;
										i2 = list[n].sourceIndex % no_of_monomials_n2;

										x = monomial_info->monomial_list[i1].ix;
										y = monomial_info->monomial_list[i1].iy;
										z = monomial_info->monomial_list[i1].iz;
										sum_n = x+y+z;

										x += monomial_info->monomial_list[i2].ix;
										y += monomial_info->monomial_list[i2].iy;
										z += monomial_info->monomial_list[i2].iz;

										if(sum_n % 2 == 1) {
											pre_factor = resultPreFactor * list[n].coeff * std::pow(inv_alpha, -list[n].a_power)*psi->coeff;
										} else {
											pre_factor = -resultPreFactor * list[n].coeff * std::pow(inv_alpha, -list[n].a_power)*psi->coeff;
										}

										vector<monomial_help_class> monomial_vector_tmp = genereate_polynom(0, x, y, z);

										for(int z = 0; z < monomial_vector_tmp.size(); z++) {
											const qmmm_real boys_factor = std::pow(-2*alpha, monomial_vector_tmp[z].length);
											monomial_vector_tmp[z].factor *= pre_factor * boys_factor;
											//calculation is multiplied with factor, if factor == 0 => then result == 0 as well
											if(fabs(monomial_vector_tmp[z].factor) != 0) {
												for(int w = 0; w < monomial_vector_tmp[z].x; w++) {
													work_list.push_back(DX);
												}
												for(int w = 0; w < monomial_vector_tmp[z].y; w++) {
													work_list.push_back(DY);
												}
												for(int w = 0; w < monomial_vector_tmp[z].z; w++) {
													work_list.push_back(DZ);
												}
												work_list.push_back(ADD); // end of monomial, use add
												work_list.push_back(monomial_vector_tmp[z].factor);
												work_list.push_back((qmmm_real)monomial_vector_tmp[z].length);
											}
										}
									}
								}
							} /* END FOR NEW PRIMS */
						}// end if number_of_new_prims > 0
					} /* END FOR NEW PRIMS */
				} /* END FOR k */
			} /* END FOR j */


			/* Anything to calculate for this index? */
			if(work_list.size() > 0) {

				clean_work_list(work_list);

				hipEventSynchronize(done[current_stream]);

				if(work_list.size() > estimated_max_size_work_list){
					estimated_max_size_work_list = work_list.size();
					cuda_success |= (unsigned int) hipFree(d_work_list);
					cuda_success |= (unsigned int) hipMalloc(&d_work_list, sizeof(qmmm_real) * estimated_max_size_work_list*nr_streams);
					cuda_success |= (unsigned int) hipHostFree(h_work_list);
					cuda_success |= (unsigned int) hipHostMalloc(&h_work_list, sizeof(qmmm_real)*estimated_max_size_work_list*nr_streams);
				}

				if(is_case_0(work_list)) {

					compute_V_matrix_full_gpu_kernel_0<<<blocksPerGrid, threadsPerBlock, 0, stream[current_stream]>>>(work_list[1], // center coords
							work_list[2],
							work_list[3],
							work_list[5],  // alpha
							work_list[8],  // boys_pre_factor
							work_list[10], // factor
							d_partial_result);
				} else {
					memcpy(h_work_list+current_stream*estimated_max_size_work_list, (void*)&work_list[0],sizeof(qmmm_real)*work_list.size());
					cuda_success |= (unsigned int) hipMemcpyAsync(d_work_list+current_stream*estimated_max_size_work_list, h_work_list+current_stream*estimated_max_size_work_list, sizeof(qmmm_real) * work_list.size(), hipMemcpyHostToDevice,stream[current_stream]);

					hipEventRecord(done[current_stream],stream[current_stream]);

					compute_V_matrix_full_gpu_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream[current_stream]>>>(d_partial_result,
							d_work_list+current_stream*estimated_max_size_work_list,
							work_list.size());
				}

				if(blocksInReduction == 1){
					reduce(num_mm_atoms, threadsInReduction, blocksInReduction, d_partial_result, d_result+i,stream[current_stream]);
				} else {
					reduce(num_mm_atoms, threadsInReduction, blocksInReduction, d_partial_result, d_partial_result,stream[current_stream]);
					reduce(blocksInReduction,threadsInReduction2,blocksInReduction2, d_partial_result,d_result+i,stream[current_stream]);
				}

				current_stream = (current_stream + 1)%nr_streams;
			}
		} // END ITERATIONS (mu & nu), END PARALLEL LOOP

		//TODO Only memcpy the number of iterations this thread has done
		cuda_success |= (unsigned int) hipMemcpy(h_result, d_result, sizeof(qmmm_real) * iterations, hipMemcpyDeviceToHost);

		cout << "Device " << dev_ID << " did " << iteration_indexes.size() << " iterations ("<< 100 * iteration_indexes.size() / iterations << "%)"<< endl;

		for(int j=0; j < iteration_indexes.size(); j++) {
			const int i = iteration_indexes[j];
			const int mu = sqrt(2*i + 0.25) - 0.5;
			const int nu = i - (mu * (mu + 1)) / 2;
			const int result_index = mu*number_of_basis_functions + nu;
			result[result_index] = h_result[i];
		}

		cuda_success |= (unsigned int) hipFree(d_cuda_atoms);
		cuda_success |= (unsigned int) hipFree(d_work_list);
		cuda_success |= (unsigned int) hipFree(d_partial_result);
		cuda_success |= (unsigned int) hipFree(d_cuda_boys_list);
		cuda_success |= (unsigned int) hipFree(d_result);
		cuda_success |= (unsigned int) hipHostFree(h_work_list);
		for(int i=0; i < nr_streams; i++)
			cuda_success |= (unsigned int) hipEventDestroy(done[i]);
		for (int i = 0; i < nr_streams; ++i)
			hipStreamDestroy(stream[i]);
		cuda_success |= (unsigned int) hipDeviceReset();

		if(cuda_success != 0) {
			cout << "[Device " << dev_ID << "] CUDA error = " << cuda_success << endl;
		}
		free(partial_result);
		free(partial_reduce_result);
		free(h_result);
#ifdef _OPENMP
	} // end parallel section
#endif


	// copy values to the other triangle
	for(int mu = 0; mu < number_of_basis_functions; mu++)
		for(int nu = mu+1; nu < number_of_basis_functions; nu++)
			result[mu*number_of_basis_functions+nu] = result[nu*number_of_basis_functions+mu];

#ifdef _OPENMP
	time_stop = omp_get_wtime();
	std::cout << "Execution time of compute_V_matrix_gpu = " << time_stop - time_start << std::endl;
#endif

	free(cuda_atoms);
	free(cuda_boys_list);
	delete hermite_conversion_info;

	//std::cout <<"Exit compute_V_matrix_gpu" << std::endl;

	return 0;
}
