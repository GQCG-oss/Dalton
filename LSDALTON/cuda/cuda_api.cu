#include <hip/hip_runtime.h>


extern "C" {
void get_dev_mem(size_t& total, size_t& free) 
{
    hipMemGetInfo(&free, &total);
}
}
